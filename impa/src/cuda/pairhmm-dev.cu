#include "hip/hip_runtime.h"
#include <cmath>
#include <cassert>
#include <string>
#include <iostream>
#include <iomanip>
#include <vector>

#include "cuPrintf.cu"

using std::string;
using std::vector;
using std::cin;
using std::cout;
using std::cerr;

#define TID threadIdx.x
#define NOT_DONE 0
#define DONE 1
#define GAP_TO(sz, n) ((n - ((sz) % n)) % n)
#define FIRST_THREAD (TID==0)
#define PH2PR(n) (pow(NUM(10.0), -NUM(n) / NUM(10.0)))

#define MAX_HAPLEN 3072
#define NBLOCKS 50
#define NUM_OF_TESTCASES_PER_ITERATION 10000

template<class T> __device__ static inline T INITIAL_CONSTANT();
template<> __device__ static inline float INITIAL_CONSTANT<float>() { return 1e32f; }
template<> __device__ static inline double INITIAL_CONSTANT<double>() { return ldexp(1.0, 1020); }
template<class T> __device__ static inline T MIN_ACCEPTED();
template<> __device__ static inline float MIN_ACCEPTED<float>() { return 1e-28f; }
template<> __device__ static inline double MIN_ACCEPTED<double>() { return 0.0; }

struct Pair {
	double result;
	int offset_hap, offset_rs, offset_qidc, haplen, rslen, status;
};

template <int nThreads, typename NUM> 
__global__ void compute_full_scores(char *g_chunk, int num_of_pairs, Pair *g_pair, int *g_pair_index, NUM *g_last_lines, int *g_last_lines_index) {

	// *******************************************************************************************
	// *********************************** <PERSISTENT VALUES> ***********************************

	__shared__ NUM *s_lastM;
	if (FIRST_THREAD)
		s_lastM = g_last_lines + 3 * MAX_HAPLEN * atomicAdd(g_last_lines_index, 1); // GLOBAL MEMORY ACCESS

	__syncthreads(); // ?? maybe we can remove this syncthreads by moving this computation later

	NUM *g_lastM = s_lastM;
	NUM *g_lastX = g_lastM + MAX_HAPLEN;
	NUM *g_lastY = g_lastX + MAX_HAPLEN;

	__shared__ NUM s_xp[nThreads], s_yp[nThreads], s_mp[nThreads];
	__shared__ NUM s_xpp[nThreads], s_ypp[nThreads], s_mpp[nThreads];

	// *********************************** </PERSISTENT VALUES> **********************************
	// *******************************************************************************************

	for (;;) {

		// *******************************************************************************************
		// ************************************** <PICK A PAIR> **************************************
		__shared__ Pair s_pair;
		__shared__ int s_pair_index;

		if (FIRST_THREAD) {
			s_pair_index = atomicAdd(g_pair_index, 1);
			if (s_pair_index < num_of_pairs)
				s_pair = g_pair[s_pair_index];
		}

		__syncthreads();

		int pair_index = s_pair_index;

		if (pair_index >= num_of_pairs)
			break;

		Pair pair = s_pair;

		if (pair.status == DONE)
			continue;

		// ************************************** </PICK A PAIR> *************************************
		// *******************************************************************************************

		int n_groups_of_rows = ((pair.rslen + 1) + (nThreads - 1)) / nThreads;
		for (int group_of_rows = 0; group_of_rows < n_groups_of_rows; group_of_rows++) {

			int row = group_of_rows * nThreads + TID;

			/******************************************************************************************
			*********** <SET THE VALUES THAT ARE CONSTANT DURING THE CALCULATION OF THE ROW> *********/
			char rs;
			char4 qidc;
			NUM mm, gm, mx, xx, my, yy, pq;
			if (row > 0 && row <= pair.rslen) {
				rs = (g_chunk + pair.offset_rs)[row-1]; // GLOBAL MEMORY ACCESS
				qidc = reinterpret_cast<char4 *>(g_chunk + pair.offset_qidc)[row-1]; // GLOBAL MEMORY ACCESS //?? each byte in the qidc char4 should be masked with 127 during chunk creation
				mm = NUM(1.0) - PH2PR(qidc.y) * PH2PR(qidc.z);
				gm = NUM(1.0) - PH2PR(qidc.w);
				mx = PH2PR(qidc.y);
				xx = PH2PR(qidc.w);
				my = (row == pair.rslen) ? NUM(1.0) : PH2PR(qidc.z);
				yy = (row == pair.rslen) ? NUM(1.0) : PH2PR(qidc.w);
				pq = PH2PR(qidc.x);
			}

			NUM k = INITIAL_CONSTANT<NUM>() / pair.haplen;

			/********** </SET THE VALUES THAT ARE CONSTANT DURING THE CALCULATION OF THE ROW> **********
			*******************************************************************************************/

			// diagonal 0
			s_mpp[TID] = NUM(0.0);
			s_xpp[TID] = NUM(0.0);
			s_ypp[TID] = NUM(0.0);

			//diagonal 1
			char hap;
			NUM m, x, y, sum_m_x = NUM(0.0), coef;

			s_mp[TID] = NUM(0.0);
			s_xp[TID] = NUM(0.0);
			s_yp[TID] = NUM(0.0);

			/*
				The purpose of the code inside the following "if" is:

				* = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = *
				* to properly set the values of the first two diagonals. There two situations *
				* that we need to take care:                                                  *
				* = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = *

				a) the group of rows (i.e.: the horizontal _tile_) is the first one
				b) the group of rows is not the first one.

				In case (a), the values of x, y and m should be initialized with 0 for all
				rows but the row 0, in which m and x should be 0 and y should be equal to
				the constant "k" (defined inside the if).

				In case (b), the only cell with values different to 0 is the cell located
				at the first row of the group of rows (i.e.: the row traversed by the thread 0)
				and the column 1 (i.e.: the first cell of the diagonal 1).

				The values of x, y and m at that cell will depend on the last values computed
				in the previous group of rows.
			*/	

			if (FIRST_THREAD) 
			{
				if (group_of_rows == 0) // case (a): 
				{
					s_ypp[0] = s_yp[0] = k;
				}
				else // case (b): just need to compute the values of m, x and y at [0][1]. NEED: the character [0] of the haplotype.
				{
					hap = g_chunk[pair.offset_hap]; // GLOBAL MEMORY ACCESS
					coef = pq;
					if (rs == hap || rs == 'N' || hap == 'N')
						coef = NUM(1.0) - coef;

					NUM m_up_left = g_lastM[0]; // lastM, lastX and lastY have (pair.haplen+1) cells, corresponding to all the columns of the matrix. The first column represents the empty string.
					NUM x_up_left = g_lastX[0];
					NUM y_up_left = g_lastY[0];
					NUM m_up = g_lastM[1]; 
					NUM x_up = g_lastX[1];

					m = coef * (m_up_left * mm + x_up_left * gm + y_up_left * gm);
					x = m_up * mx + x_up * xx;
					y = NUM(0.0);

					s_xp[0] = x;
					s_yp[0] = y;
					s_mp[0] = m;

					sum_m_x += m + x;
				}
			} // end of "if (FIRST_THREAD) 

			int num_of_diagonals = (pair.haplen+1) + (pair.rslen+1) - 1;
			for (int d = 2; d < num_of_diagonals; d++) 
			{
				__syncthreads();

				int col = d - TID;
				hap = 0;
				NUM m_up_left = NUM(0.0), x_up_left = NUM(0.0), y_up_left = NUM(0.0);
				NUM m_up = NUM(0.0), x_up = NUM(0.0);
				NUM m_left = s_mp[0], y_left = s_yp[0];
				
				if (FIRST_THREAD) {
					if (group_of_rows == 0) {
						yy = NUM(1.0);
						my = NUM(0.0);
					}
					else {
						if (col <= pair.haplen) {
							hap = g_chunk[pair.offset_hap + (col-1)]; // GLOBAL MEMORY ACCESS
							m_up_left = g_lastM[col-1]; // GLOBAL MEMORY ACCESS
							x_up_left = g_lastX[col-1]; // GLOBAL MEMORY ACCESS
							y_up_left = g_lastY[col-1]; // GLOBAL MEMORY ACCESS
							m_up = g_lastM[col]; // GLOBAL MEMORY ACCESS
							x_up = g_lastX[col]; // GLOBAL MEMORY ACCESS
							m_left = s_mp[0];
							y_left = s_yp[0];
						}
					}
				} 
				else {
					if (col > 0 && col <= pair.haplen) // ?? haplotype should be padded and this "if", removed.
						hap = g_chunk[pair.offset_hap + (col-1)]; // GLOBAL MEMORY ACCESS
					m_up_left = s_mpp[TID-1]; 
					x_up_left = s_xpp[TID-1];
					y_up_left = s_ypp[TID-1];
					m_up = s_mp[TID-1];
					x_up = s_xp[TID-1];
					m_left = s_mp[TID];
					y_left = s_yp[TID];
				}

				coef = pq;
				if (rs == hap || rs == 'N' || hap == 'N')
					coef = NUM(1.0) - coef;
				m = coef * (m_up_left * mm + x_up_left * gm + y_up_left * gm);
				x = m_up * mx + x_up * xx;
				y = m_left * my + y_left * yy;


				if ((TID == (nThreads-1)) && (group_of_rows < n_groups_of_rows - 1) && (col >= 0) && (col <= pair.haplen))
				{
					g_lastX[col] = x; // GLOBAL MEMORY ACCESS
					g_lastY[col] = y; // GLOBAL MEMORY ACCESS
					g_lastM[col] = m; // GLOBAL MEMORY ACCESS
				}

				if (/*col >= 0 && */col <= pair.haplen)
					sum_m_x += m + x;

				s_xpp[TID] = s_xp[TID];	
				s_ypp[TID] = s_yp[TID];	
				s_mpp[TID] = s_mp[TID];
				s_xp[TID] = x; 
				s_yp[TID] = y; 
				s_mp[TID] = m;

//				__syncthreads();
			} // end of the for (d = 2 to {number of diagonals}) 

			if (row == pair.rslen) 
			{
				pair.result = double(log10(sum_m_x) - log10(INITIAL_CONSTANT<NUM>()));
				pair.status = (sum_m_x >= MIN_ACCEPTED<NUM>()) ? DONE : NOT_DONE;
				g_pair[s_pair_index] = pair; // GLOBAL MEMORY ACCESS
			}
		} // end of the for (group_of rows = ...)
	} // end of the for (;;)

	return;
}

int create_chunk(vector<Pair> &pairs, string &chunk)
{
	vector<Pair>().swap(pairs); 
	string("").swap(chunk);

	int current_offset = 0;
	std::string hap, rs, q, i, d, c, i1, i2;
	while (pairs.size() < NUM_OF_TESTCASES_PER_ITERATION && (cin >> hap >> rs >> q >> i >> d >> c >> i1 >> i2).good())
	{
		string tchunk("");
		int tchunk_sz = hap.size() + 1 + rs.size() + 1; 
		tchunk = hap + string(1, '\0') + rs + string(1, '\0');
		tchunk += string(GAP_TO(tchunk_sz, 4), '\0');
		tchunk_sz += GAP_TO(tchunk_sz, 4);

		for (int x = 0; x < rs.size(); x++)
		{
			char tq = q[x] - 33; if (tq < 6) tq = 6; tq = (tq & 127);
			char ti = i[x] - 33;
			char td = d[x] - 33;
			char tc = c[x] - 33;
			tchunk += string(1, tq) + string(1, ti) + string(1, td) + string(1, tc);
			tchunk_sz += 4;
		}
		tchunk += string(4, '\0');
		tchunk_sz += 4;

	  assert(tchunk.size() == tchunk_sz);
		tchunk += string(GAP_TO(tchunk.size(), 128), '\0');
		tchunk_sz += GAP_TO(tchunk_sz, 128);

		Pair p;
		p.status = NOT_DONE;
		p.result = 0.0;
		p.offset_hap = current_offset;
		p.offset_rs = current_offset + (hap.size()+1);
		int sz_hap_rs = (hap.size()+1) + (rs.size()+1);
		p.offset_qidc = current_offset +  sz_hap_rs + GAP_TO(sz_hap_rs, 4);
		p.haplen = hap.size();
		p.rslen = rs.size();

		chunk += tchunk;
		pairs.push_back(p);
	  assert(tchunk_sz == tchunk.size());

		current_offset += tchunk.size();
	}

	return pairs.size();
}

int main(void)
{
	dim3 gridDim(NBLOCKS);
	dim3 blockDim(NTHREADS);

	string chunk;
	vector<Pair> pairs;

	while (create_chunk(pairs, chunk))
	{
		char *g_chunk;
		int padd_sz = 128;
		assert( hipMalloc(&g_chunk, chunk.size() + 2 * padd_sz) == hipSuccess);

		assert( hipMemset(g_chunk, 0, padd_sz) == hipSuccess);
		assert( hipMemset(g_chunk + padd_sz + chunk.size(), 0, padd_sz) == hipSuccess);
		assert( hipMemcpy(g_chunk + padd_sz, chunk.c_str(), chunk.size(), hipMemcpyHostToDevice) == hipSuccess);
		g_chunk += padd_sz;

		Pair *g_pair;
		assert( hipMalloc(&g_pair, pairs.size() * sizeof(Pair)) == hipSuccess);
		assert( hipMemcpy(g_pair, &(pairs[0]), pairs.size() * sizeof(Pair), hipMemcpyHostToDevice) == hipSuccess);

		int *g_pair_index, *g_last_lines_index;
		assert( hipMalloc(&g_pair_index, sizeof(int)) == hipSuccess);
		assert( hipMalloc(&g_last_lines_index, sizeof(int)) == hipSuccess);

		cudaPrintfInit();

		void *g_last_lines;
		assert( hipMalloc(&g_last_lines, NBLOCKS * MAX_HAPLEN * 3 * sizeof(double)) == hipSuccess);

		assert( hipMemset(g_pair_index, 0, sizeof(int)) == hipSuccess);
		assert( hipMemset(g_last_lines_index, 0, sizeof(int)) == hipSuccess);
		compute_full_scores<32, float><<<gridDim, blockDim>>>(g_chunk, pairs.size(), g_pair, g_pair_index, reinterpret_cast<float *>(g_last_lines), g_last_lines_index);

		assert( hipMemset(g_pair_index, 0, sizeof(int)) == hipSuccess);
		assert( hipMemset(g_last_lines_index, 0, sizeof(int)) == hipSuccess);
		compute_full_scores<32, double><<<gridDim, blockDim>>>(g_chunk, pairs.size(), g_pair, g_pair_index, reinterpret_cast<double *>(g_last_lines), g_last_lines_index);

		cudaPrintfDisplay();
		cudaPrintfEnd();

		assert( hipMemcpy(&(pairs[0]), g_pair, pairs.size() * sizeof(Pair), hipMemcpyDeviceToHost) == hipSuccess);

	
		cout << std::setprecision(16);
		for (int p = 0; p < pairs.size(); p++)
			cout << pairs[p].result << "\n";

		assert( hipFree(g_last_lines) == hipSuccess);
		assert( hipFree(g_last_lines_index) == hipSuccess);
		assert( hipFree(g_pair) == hipSuccess);
		assert( hipFree(g_pair_index) == hipSuccess);
		assert( hipFree(g_chunk-padd_sz) == hipSuccess);
	}

	return 0;
}


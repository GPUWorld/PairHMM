#include "hip/hip_runtime.h"
#include <cmath>
#include <cassert>
#include <string>
#include <iostream>
#include <iomanip>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <sys/time.h>

//#include "cuPrintf.cu"

using std::string;
using std::vector;
using std::cin;
using std::cout;
using std::cerr;

#define NOT_DONE 0
#define DONE 1
#define GAP_TO(sz, n) ((n - ((sz) % n)) % n)
#define PH2PR(n) (pow(NUM(10.0), -NUM(n) / NUM(10.0)))

#define MAX_HAPLEN 3072
#define NBLOCKS 150
#define NUM_OF_TESTCASES_PER_ITERATION 10000

template<class T> __device__ static inline T INITIAL_CONSTANT();
template<> __device__ static inline float INITIAL_CONSTANT<float>() { return 1e32f; }
template<> __device__ static inline double INITIAL_CONSTANT<double>() { return ldexp(1.0, 1020); }
template<class T> __device__ static inline T MIN_ACCEPTED();
template<> __device__ static inline float MIN_ACCEPTED<float>() { return 1e-28f; }
template<> __device__ static inline double MIN_ACCEPTED<double>() { return 0.0; }

struct Pair 
{
	double result;
	int offset_hap, offset_rs, offset_qidc, haplen, rslen, status;
};

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		cerr << "Error code: " << code << "\n";
		cerr << "Description: " << hipGetErrorString(code) << "\n";
		cerr << "Line: " << line << "\n";
		exit(code);
	}
}

template <int nThreads, typename NUM> 
__global__ void compute_full_scores(char *g_chunk, int num_of_pairs, Pair *g_pair, int *g_pair_index, NUM *g_last_lines)
{

	// *******************************************************************************************
	// *********************************** <PERSISTENT VALUES> ***********************************

	NUM *g_lastM = g_last_lines + 3 * MAX_HAPLEN * blockIdx.x;
	NUM *g_lastX = g_lastM + MAX_HAPLEN;
	NUM *g_lastY = g_lastX + MAX_HAPLEN;

	__shared__ NUM s_xp[nThreads], s_yp[nThreads], s_mp[nThreads];
	__shared__ NUM s_xpp[nThreads], s_ypp[nThreads], s_mpp[nThreads];

	// *********************************** </PERSISTENT VALUES> **********************************
	// *******************************************************************************************

	for (;;)
	{
		// *******************************************************************************************
		// ************************************** <PICK A PAIR> **************************************
		__syncthreads();

		__shared__ Pair s_pair;
		__shared__ int s_pair_index;

		if (threadIdx.x == 0)
		{
			s_pair_index = atomicAdd(g_pair_index, 1);
			if (s_pair_index < num_of_pairs)
				s_pair = g_pair[s_pair_index];
		}

		__syncthreads();

		int pair_index = s_pair_index;

		if (pair_index >= num_of_pairs)
			break;

		Pair pair = s_pair;

		if (pair.status == DONE)
			continue;

		// ************************************** </PICK A PAIR> *************************************
		// *******************************************************************************************

		int n_groups_of_rows = ((pair.rslen + 1) + (nThreads - 1)) / nThreads;
		for (int group_of_rows = 0; group_of_rows < n_groups_of_rows; group_of_rows++)
		{
			__syncthreads();

			int row = group_of_rows * nThreads + threadIdx.x;

			/******************************************************************************************
			*********** <SET THE VALUES THAT ARE CONSTANT DURING THE CALCULATION OF THE ROW> *********/
			char rs;
			char4 qidc;
			NUM mm, gm, mx, xx, my, yy, pq;
			if (row > 0 && row <= pair.rslen)
			{
				rs = (g_chunk + pair.offset_rs)[row-1]; // GLOBAL MEMORY ACCESS
				qidc = reinterpret_cast<char4 *>(g_chunk + pair.offset_qidc)[row-1]; // GLOBAL MEMORY ACCESS //?? each byte in the qidc char4 should be masked with 127 during chunk creation
				mm = NUM(1.0) - PH2PR(qidc.y) * PH2PR(qidc.z);
				gm = NUM(1.0) - PH2PR(qidc.w);
				mx = PH2PR(qidc.y);
				xx = PH2PR(qidc.w);
				my = (row == pair.rslen) ? NUM(1.0) : PH2PR(qidc.z);
				yy = (row == pair.rslen) ? NUM(1.0) : PH2PR(qidc.w);
				pq = PH2PR(qidc.x);
			}

			NUM k = INITIAL_CONSTANT<NUM>() / pair.haplen;

			/********** </SET THE VALUES THAT ARE CONSTANT DURING THE CALCULATION OF THE ROW> **********
			*******************************************************************************************/

			// diagonal 0
			s_mpp[threadIdx.x] = NUM(0.0);
			s_xpp[threadIdx.x] = NUM(0.0);
			s_ypp[threadIdx.x] = NUM(0.0);

			//diagonal 1
			char hap;
			NUM m, x, y, sum_m_x = NUM(0.0), coef;

			s_mp[threadIdx.x] = NUM(0.0);
			s_xp[threadIdx.x] = NUM(0.0);
			s_yp[threadIdx.x] = NUM(0.0);

			__syncthreads();

			/*
				The purpose of the code inside the following "if" is:

				* = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = *
				* to properly set the values of the first two diagonals. There two situations *
				* that we need to take care:                                                  *
				* = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = = *

				a) the group of rows (i.e.: the horizontal _tile_) is the first one
				b) the group of rows is not the first one.

				In case (a), the values of x, y and m should be initialized with 0 for all
				rows but the row 0, in which m and x should be 0 and y should be equal to
				the constant "k" (defined inside the if).

				In case (b), the only cell with values different to 0 is the cell located
				at the first row of the group of rows (i.e.: the row traversed by the thread 0)
				and the column 1 (i.e.: the first cell of the diagonal 1).

				The values of x, y and m at that cell will depend on the last values computed
				in the previous group of rows.
			*/	

			if (threadIdx.x == 0) 
			{
				if (group_of_rows == 0) // case (a): 
				{
					s_ypp[0] = s_yp[0] = k;
				}
				else // case (b): just need to compute the values of m, x and y at [0][1]. NEED: the character [0] of the haplotype.
				{
					hap = g_chunk[pair.offset_hap]; // GLOBAL MEMORY ACCESS
					coef = pq;
					if (rs == hap || rs == 'N' || hap == 'N')
						coef = NUM(1.0) - coef;

					NUM m_up_left = g_lastM[0]; // lastM, lastX and lastY have (pair.haplen+1) cells, corresponding to all the columns of the matrix. The first column represents the empty string.
					NUM x_up_left = g_lastX[0];
					NUM y_up_left = g_lastY[0];
					NUM m_up = g_lastM[1]; 
					NUM x_up = g_lastX[1];

					m = coef * (m_up_left * mm + x_up_left * gm + y_up_left * gm);
					x = m_up * mx + x_up * xx;
					y = NUM(0.0);

					s_xp[0] = x;
					s_yp[0] = y;
					s_mp[0] = m;

					sum_m_x += m + x;
				}
			} // end of "if (threadIdx.x==0) 
		
			__syncthreads();

			int num_of_diagonals = (pair.haplen+1) + (pair.rslen+1) - 1;
			for (int d = 2; d < num_of_diagonals; d++) 
			{
				__syncthreads();

				int col = d - threadIdx.x;
				hap = 0;
				NUM m_up_left = NUM(0.0), x_up_left = NUM(0.0), y_up_left = NUM(0.0);
				NUM m_up = NUM(0.0), x_up = NUM(0.0);
				NUM m_left = s_mp[0], y_left = s_yp[0];

//__syncthreads();
				if (threadIdx.x==0)
				{
					if (group_of_rows == 0)
					{
						yy = NUM(1.0);
						my = NUM(0.0);
					}
					else
					{
						if (col > 0 && col <= pair.haplen)
						{
							hap = g_chunk[pair.offset_hap + (col-1)]; // GLOBAL MEMORY ACCESS
							m_up_left = g_lastM[col-1]; // GLOBAL MEMORY ACCESS
							x_up_left = g_lastX[col-1]; // GLOBAL MEMORY ACCESS
							y_up_left = g_lastY[col-1]; // GLOBAL MEMORY ACCESS
							m_up = g_lastM[col]; // GLOBAL MEMORY ACCESS
							x_up = g_lastX[col]; // GLOBAL MEMORY ACCESS
							m_left = s_mp[0];
							y_left = s_yp[0];
						}
					}
				} 
				else
				{
					if (col > 0 && col <= pair.haplen) // ?? haplotype should be padded and this "if", removed.
						hap = g_chunk[pair.offset_hap + (col-1)]; // GLOBAL MEMORY ACCESS
					m_up_left = s_mpp[threadIdx.x-1]; 
					x_up_left = s_xpp[threadIdx.x-1];
					y_up_left = s_ypp[threadIdx.x-1];
					m_up = s_mp[threadIdx.x-1];
					x_up = s_xp[threadIdx.x-1];
					m_left = s_mp[threadIdx.x];
					y_left = s_yp[threadIdx.x];
				}

				coef = pq;
				if (rs == hap || rs == 'N' || hap == 'N')
					coef = NUM(1.0) - coef;
				m = coef * (m_up_left * mm + x_up_left * gm + y_up_left * gm);
				x = m_up * mx + x_up * xx;
				y = m_left * my + y_left * yy;

				if ((threadIdx.x == (nThreads-1)) && (group_of_rows < n_groups_of_rows - 1) && (col >= 0) && (col <= pair.haplen))
				{
					g_lastX[col] = x; // GLOBAL MEMORY ACCESS
					g_lastY[col] = y; // GLOBAL MEMORY ACCESS
					g_lastM[col] = m; // GLOBAL MEMORY ACCESS
				}

				__syncthreads(); // ?? this __syncthreads() has shown to be necessary, but it is not clear to me the reason. TO DO: Understand deeply what's happening in here. This code works, but it is mandatory to know why removing this __syncthreads() makes the program fail.

				if (col >= 0 && col <= pair.haplen)
					sum_m_x += m + x;

				s_xpp[threadIdx.x] = s_xp[threadIdx.x];	
				s_ypp[threadIdx.x] = s_yp[threadIdx.x];	
				s_mpp[threadIdx.x] = s_mp[threadIdx.x];
				s_xp[threadIdx.x] = x; 
				s_yp[threadIdx.x] = y; 
				s_mp[threadIdx.x] = m;
			} // end of the for (d = 2 to {number of diagonals}). Always followed by __syncthreads();

			__syncthreads();

			if (row == pair.rslen) 
			{
				pair.result = double(log10(sum_m_x) - log10(INITIAL_CONSTANT<NUM>()));
				pair.status = (sum_m_x >= MIN_ACCEPTED<NUM>()) ? DONE : NOT_DONE;
				g_pair[s_pair_index] = pair; // GLOBAL MEMORY ACCESS
			}
		} // end of the for (group_of rows = ...). Always followed by __syncthreads();

		__syncthreads();

	} // end of the for (;;)

	return;
}

int create_chunk(vector<Pair> &pairs, string &chunk)
{
	vector<Pair>().swap(pairs); 
	chunk.clear();

	int current_offset = 0;
	std::string hap, rs, q, i, d, c, i1, i2, tchunk;
	while (pairs.size() < NUM_OF_TESTCASES_PER_ITERATION && (cin >> hap >> rs >> q >> i >> d >> c >> i1 >> i2).good())
	{
		tchunk.clear();
		tchunk = hap + string(1, '\0') + rs + string(1, '\0');
		tchunk += string(GAP_TO(tchunk.size()/*tchunk_sz*/, 4), '\0');

		for (int x = 0; x < rs.size(); x++)
		{
			char tq = q[x] - 33; if (tq < 6) tq = 6; tq = (tq & 127);
			char ti = (i[x] - 33) & 127;
			char td = (d[x] - 33) & 127;
			char tc = (c[x] - 33) & 127;
			tchunk += string(1, tq) + string(1, ti) + string(1, td) + string(1, tc);
		}
		tchunk += string(4, '\0');
		tchunk += string(GAP_TO(tchunk.size(), 128), '\0');

		Pair p;
		p.status = NOT_DONE;
		p.result = 0.0;
		p.offset_hap = current_offset;
		p.offset_rs = current_offset + (hap.size()+1);
		int sz_hap_rs = (hap.size()+1) + (rs.size()+1);
		p.offset_qidc = current_offset +  sz_hap_rs + GAP_TO(sz_hap_rs, 4);
		p.haplen = hap.size();
		p.rslen = rs.size();

		chunk += tchunk;
		pairs.push_back(p);

		current_offset += tchunk.size();
		hap.clear();
		rs.clear();
		q.clear();
		i.clear();
		d.clear();
		c.clear();
	}

	return pairs.size();
}

class Timing
{
public:
	Timing(string t) : st(now()), tot(0.0), title(t)
	{
	}

	~Timing()
	{
		std::cerr << title << tot << " seconds\n";
	}

	void start()
	{
		st = now();
	}

	void acc()
	{
		tot += (now() - st);
	}

private:
	static double now()
	{
		struct timeval v;
		gettimeofday(&v, (struct timezone *) NULL);
		return v.tv_sec + v.tv_usec/1.0e6;
	}

	double st, tot;
	string title;
};

int main(void)
{
	Timing TotalTime(string("TOTAL: "));
	Timing ComputationTime(string("COMPUTATION: "));

	dim3 gridDim(NBLOCKS);
	dim3 blockDim(NTHREADS);

	string chunk;
	vector<Pair> pairs;

	while (create_chunk(pairs, chunk))
	{
		ComputationTime.start();

		char *g_chunk;
		int padd_sz = 128;
		gpuErrchk( hipMalloc(&g_chunk, chunk.size() + 2 * padd_sz) );

		gpuErrchk( hipMemset(g_chunk, 0, padd_sz) );
		gpuErrchk( hipMemset(g_chunk + padd_sz + chunk.size(), 0, padd_sz) );
		gpuErrchk( hipMemcpy(g_chunk + padd_sz, chunk.c_str(), chunk.size(), hipMemcpyHostToDevice) );
		g_chunk += padd_sz;

		Pair *g_pair;
		gpuErrchk( hipMalloc(&g_pair, pairs.size() * sizeof(Pair)) );
		gpuErrchk( hipMemcpy(g_pair, &(pairs[0]), pairs.size() * sizeof(Pair), hipMemcpyHostToDevice) );

		int *g_pair_index;
		gpuErrchk( hipMalloc(&g_pair_index, sizeof(int)) );

//cudaPrintfInit();

		void *g_last_lines;
		gpuErrchk( hipMalloc(&g_last_lines, NBLOCKS * MAX_HAPLEN * 3 * sizeof(double)) );

		gpuErrchk( hipMemset(g_pair_index, 0, sizeof(int)) );
		compute_full_scores<NTHREADS, float><<<gridDim, blockDim>>>(g_chunk, pairs.size(), g_pair, g_pair_index, reinterpret_cast<float *>(g_last_lines));

hipDeviceSynchronize(); // agregado por si acaso... no es esto...
		gpuErrchk( hipMemset(g_pair_index, 0, sizeof(int)) );
		compute_full_scores<NTHREADS, double><<<gridDim, blockDim>>>(g_chunk, pairs.size(), g_pair, g_pair_index, reinterpret_cast<double *>(g_last_lines));

//cudaPrintfDisplay();
//cudaPrintfEnd();

		gpuErrchk( hipMemcpy(&(pairs[0]), g_pair, pairs.size() * sizeof(Pair), hipMemcpyDeviceToHost) );

		cout << std::setprecision(16);
		for (int p = 0; p < pairs.size(); p++)
			cout << pairs[p].result << "\n";

		gpuErrchk( hipFree(g_last_lines) );
		gpuErrchk( hipFree(g_pair) );
		gpuErrchk( hipFree(g_pair_index) );
		gpuErrchk( hipFree(g_chunk-padd_sz) );

		ComputationTime.acc();
	}

	TotalTime.acc();

	return 0;
}


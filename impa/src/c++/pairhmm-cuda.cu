#include "hip/hip_runtime.h"
#include <cstdio>
#include <string>
#include <cstring>
#include <cstdlib>
#include <cmath>
#include <iostream>
#include <fstream>

#include "input.h"
#include "compute_gpu.h"

#include "timing.h"
//#include "read.h"

template<class T>
struct Context{};

template<>
struct Context<double>
{
	Context()
	{
		for (int x = 0; x < 128; x++)
			ph2pr[x] = pow(10.0, -((double)x) / 10.0);

		INITIAL_CONSTANT = ldexp(1.0, 1020.0);
		LOG10_INITIAL_CONSTANT = log10(INITIAL_CONSTANT);
		RESULT_THRESHOLD = 0.0;
	}

	double LOG10(double v){ return log10(v); }

	static double _(double n){ return n; }
	static double _(float n){ return ((double) n); }
	double ph2pr[128];
	double INITIAL_CONSTANT;
	double LOG10_INITIAL_CONSTANT;
	double RESULT_THRESHOLD;
};

template<>
struct Context<float>
{
	Context()
	{
		for (int x = 0; x < 128; x++)
			ph2pr[x] = powf(10.f, -((float)x) / 10.f);

		INITIAL_CONSTANT = ldexpf(1.f, 120.f);
		LOG10_INITIAL_CONSTANT = log10f(INITIAL_CONSTANT);
		RESULT_THRESHOLD = ldexpf(1.f, -110.f);
	}

	float LOG10(float v){ return log10f(v); }

	static float _(double n){ return ((float) n); }
	static float _(float n){ return n; }
	float ph2pr[128];
	float INITIAL_CONSTANT;
	float LOG10_INITIAL_CONSTANT;
	float RESULT_THRESHOLD;
};
template<class T>
T ph2pr(int);
template <>
float ph2pr<float>(int x) {
			return powf(10.f, -((float)x) / 10.f);
}
template <>
double ph2pr<double>(int x) {
			return powf(10.0, -((double)x) / 10.0);
}

template<class NUMBER>
int tc2gmem(GPUmem<NUMBER>& gmem, testcase* tc, int index)
{
	int ROWS = tc->rslen + 1;
	int COLS = tc->haplen + 1;

   if (ROWS==1 || COLS==1) return 0;
#ifdef __NO_PREPACK
   extract_tc(gmem.M,gmem.X,gmem.Y,gmem.p+gmem.offset[index].x*6,gmem.n+gmem.offset[index].x,
              gmem.q+gmem.offset[index].x,tc);
#else
   memcpy(gmem.n+gmem.offset[index].x+1, tc->n_new, sizeof(int)*ROWS);
#endif
   //TODO check data sizes first
   //      return error if we're out of space
   memcpy(gmem.rs+gmem.offset[index].x, tc->rs, sizeof(char)*ROWS);
   memcpy(gmem.hap+gmem.offset[index].y, tc->hap, sizeof(char)*COLS);
   gmem.index++;
   return 0;
}
template<class NUMBER>
void extract_tc(NUMBER* M_in, NUMBER* X_in, NUMBER* Y_in, 
                NUMBER* p_in, int* n_new, NUMBER* q_new, testcase* tc)
{
	int ROWS = tc->rslen + 1;
   int r;

	for (r = 1; r < ROWS; r++)
	{
		int _i = tc->i[r-1] & 127;
		int _d = tc->d[r-1] & 127;
		int _c = tc->c[r-1] & 127;
      int _q = tc->q[r-1] & 127;
      n_new[r]=_i+128*_d+128*128*_c+128*128*128*_q;
	}

}

template<class NUMBER>
NUMBER compute_full_prob(testcase *tc, NUMBER *before_last_log = NULL)
{
	int r, c;
	int ROWS = tc->rslen + 1;
	int COLS = tc->haplen + 1;
	NUMBER INITIAL_CONSTANT = ldexp(1.0, 1020.0);
	NUMBER LOG10_INITIAL_CONSTANT = log10(INITIAL_CONSTANT);


	NUMBER M[ROWS][COLS];
	NUMBER X[ROWS][COLS];
	NUMBER Y[ROWS][COLS];
	NUMBER p[ROWS][6];

	p[0][MM] = 0.0;
	p[0][GapM] = 0.0;
	p[0][MX] = 0.0;
	p[0][XX] = 0.0;
	p[0][MY] = 0.0;
	p[0][YY] = 0.0;
	for (r = 1; r < ROWS; r++)
	{
		int _i = tc->i[r-1] & 127;
		int _d = tc->d[r-1] & 127;
		int _c = tc->c[r-1] & 127;
		p[r][MM] = 1.0 - ph2pr<NUMBER>((_i + _d) & 127);
		p[r][GapM] = 1.0 - ph2pr<NUMBER>(_c);
		p[r][MX] = ph2pr<NUMBER>(_i);
		p[r][XX] = ph2pr<NUMBER>(_c);
		p[r][MY] = (r == ROWS - 1) ? 1.0 : ph2pr<NUMBER>(_d);
		p[r][YY] = (r == ROWS - 1) ? 1.0 : ph2pr<NUMBER>(_c);
	}

	for (c = 0; c < COLS; c++)
	{
		M[0][c] = 0.0;
		X[0][c] = 0.0;
		Y[0][c] = INITIAL_CONSTANT / (tc->haplen);
	}

	for (r = 1; r < ROWS; r++)
	{
		M[r][0] = 0.0;
		X[r][0] = X[r-1][0] * p[r][XX];
		Y[r][0] = 0.0;
	}

	for (r = 1; r < ROWS; r++)
		for (c = 1; c < COLS; c++)
		{
			char _rs = tc->rs[r-1];
			char _hap = tc->hap[c-1];
			int _q = tc->q[r-1] & 127;
			NUMBER distm = ph2pr<NUMBER>(_q);
			if (_rs == _hap || _rs == 'N' || _hap == 'N')
				distm = 1.0 - distm;
			M[r][c] = distm * (M[r-1][c-1] * p[r][MM] + X[r-1][c-1] * p[r][GapM] + Y[r-1][c-1] * p[r][GapM]);
			X[r][c] = M[r-1][c] * p[r][MX] + X[r-1][c] * p[r][XX];
			Y[r][c] = M[r][c-1] * p[r][MY] + Y[r][c-1] * p[r][YY];
		}

	NUMBER result = 0.0;
	for (c = 0; c < COLS; c++)
		result += M[ROWS-1][c] + X[ROWS-1][c];

	if (before_last_log != NULL)
		*before_last_log = result;	

	return log10(result) - LOG10_INITIAL_CONSTANT;
}

int tc_comp1(const void* tc_A, const void* tc_B) {
   int rA = ((testcase*)tc_A)->rslen * ((testcase*)tc_A)->haplen; 
   int rB = ((testcase*)tc_B)->rslen * ((testcase*)tc_B)->haplen; 
   if (rA<rB) return 1;
   else return -1;
}
int tc_comp2(const void* tc_A, const void* tc_B) {
   int rA = ((testcase*)tc_A)->rslen;
   int rB = ((testcase*)tc_B)->rslen;
   if (rA<rB) return 1;
   else return -1;
}
int tc_comp_unsort(const void* tc_A, const void* tc_B) {
   return ((testcase*)tc_A)->index - ((testcase*)tc_B)->index;
}

#define round_up(A,B) (B)*((A+B-1)/(B))
//#define round_up(A,B) A

template<class NUMBER>
void compute_full_prob_multiple(double* probs, testcase *tc, int n_tc, 
                                 GPUmem<NUMBER> &gmem, NUMBER *before_last_log = NULL) {
   Context<NUMBER> ctx;
   int err;

   //if (gmem.amem == 0) debugMark<1><<<1,1>>>();
   //else debugMark<1><<<1,1,0, gmem.marker_s>>>();
   Timing All(string("compute_full_prob_multiple total :  "));
   Timing GPUAlloc(string("GPU Alloc/Free :  "));
   Timing SortTC(string("Sort :  "));
   All.start();
   SortTC.start();
#pragma omp parallel for
   for (int z=0;z<4;z++) {
      //qsort(tc+z*n_tc/4, n_tc/4, sizeof(testcase), tc_comp2);
   }
   printf("largest mat: %d x %d\n", tc[0].rslen, tc[0].haplen);
   SortTC.acc();
   GPUAlloc.start();
   if (0==n_tc) {
      fprintf(stderr, "Free GPUmem\n");
      err = GPUmemFree<NUMBER>(gmem);
      return;
   }
   if (gmem.amem==0) {
      fprintf(stderr, "Alloc GPUmem\n");
      err = GPUmemAlloc<NUMBER>(gmem);
   }
   if (err != 0) printf("Error in GPU allocation/deallocation\n");
   GPUAlloc.acc();

   Timing Staging(string("Staging :  "));
   Timing ComputeGPU(string("Compute GPU Time :  "));
   gmem.index=0;
   int total_rows=0;
   int total_cols=0;
   int total_scratch=0;
   unsigned long long total_cells = 0;
   for (int z=0;z<n_tc;z++)
   {
      //gmem.offset[z][0] = total_scratch;
      gmem.offset[z].x = total_rows;
      gmem.offset[z].y = total_cols;
      total_rows += tc[z].rslen+1;
      total_cols += tc[z].haplen+1;
      total_scratch += ((tc[z].rslen+WARP-1)/(WARP-1))*(tc[z].haplen+1);
      total_cells += tc[z].rslen*tc[z].haplen;
   }
   printf("%u cells\n", total_cells);
   //TODO clean this up!
   //gmem.offset[n_tc][0] = total_scratch;
   gmem.offset[n_tc].x = total_rows;
   gmem.offset[n_tc].y = total_cols;
   gmem.X = gmem.M + total_cols;
   gmem.d_X = gmem.d_M + total_cols;
   gmem.Y = gmem.X + total_cols;
   gmem.d_Y = gmem.d_X + total_cols;
   //q and n must be aligned to 512 bytes for transfer as textures
   gmem.q = gmem.M + round_up(3*total_cols, 512/sizeof(NUMBER));
   gmem.d_q = gmem.d_M + round_up(3*total_cols, 512/sizeof(NUMBER));
   if (((char*)gmem.d_q-(char*)gmem.d_M)%512 != 0) printf("d_q not aligned\n");
   gmem.n = (int*)(gmem.q + round_up(total_rows,512/sizeof(NUMBER)));
   gmem.d_n = (int*)(gmem.d_q + round_up(total_rows,512/sizeof(NUMBER)));
   if (((char*)gmem.d_n-(char*)gmem.d_M)%512 != 0) printf("d_n not aligned\n");
   gmem.rs = (char*)(gmem.n + total_rows*3);
   gmem.d_rs = (char*)(gmem.d_n + total_rows*3);
   gmem.hap = gmem.rs + total_rows;
   gmem.d_hap = gmem.d_rs + total_rows;
   //Make sure results and d_results are properly aligned
   //TODO put results before the input data and scratch to avoid thise complexity
   gmem.results = (NUMBER*)(gmem.hap + total_cols + sizeof(NUMBER)-(13*total_rows+total_cols)%sizeof(NUMBER));
   gmem.d_results = (NUMBER*)(gmem.d_hap + total_cols + sizeof(NUMBER)-(13*total_rows+total_cols)%sizeof(NUMBER));
   if ((char*)gmem.results+n_tc*sizeof(NUMBER)-(char*)gmem.M > gmem.totalMem) {
      fprintf(stderr, "data exceeds GPU memory. Quitting.\n");
      return;
   }
   int s=0;
   for (int start=0;start<n_tc;start+=n_tc/gmem.N_STREAMS) {
      int finish=min(start+n_tc/gmem.N_STREAMS, n_tc);
      Staging.start();
//      CPU_start<<<1,1,0,gmem.marker_s>>>();
#pragma omp parallel for shared(gmem, tc) private (z)
      for (int z=start;z<finish;z++)
      {
         err = tc2gmem<NUMBER>(gmem, &tc[z], z);
      }
//      CPU_end<<<1,1,0,gmem.marker_s>>>();
      Staging.acc();
      ComputeGPU.start();
      hipStreamSynchronize(gmem.strm[s]);
      compute_gpu_stream(gmem.offset+start, gmem.rs, gmem.hap, gmem.q, gmem.n, 
                         ctx.INITIAL_CONSTANT, finish-start, gmem, gmem.strm[s], start);
      ComputeGPU.acc();
      s++;
      s %= gmem.N_STREAMS;
   }
   ComputeGPU.start();
   for (s=0;s<gmem.N_STREAMS;s++) hipStreamSynchronize(gmem.strm[s]);
   //memcpy(probs, gmem.results, sizeof(NUMBER)*n_tc);
		#pragma omp parallel for schedule(dynamic)
   for (int z=0;z<n_tc;z++) {
      if (fabs(gmem.results[z] - 1.0000) < 0.0001)
         probs[tc[z].index] = compute_full_prob<double>(&tc[z]);
      else probs[tc[z].index] = (double)gmem.results[z];
   }
   ComputeGPU.acc();
   All.acc();
//   debugMark<2><<<1,1>>>();
} 

#ifndef DOUBLE_FLOAT
#define DOUBLE_FLOAT float
#endif
int main(int argc, char* argv[])
{
   Timing TotalTime(string("TOTAL: "));
   Timing ComputationTime(string("COMPUTATION: "));
   TotalTime.start();
	testcase *tc = new testcase[MAX_PROBS];
   int cnt=0;
   int basecnt=0;
   double *prob;
   prob = (double*)malloc(MAX_PROBS*sizeof(double));
   GPUmem<DOUBLE_FLOAT> gmem;
  
   std::ifstream infile;

   if (argc>1) {
      infile.open((char*) argv[1]);
   } 

	while (read_testcase(tc+cnt, argc>1 ? infile: std::cin) == 0)
   {
      //printf("In pairhmm-cuda: &tc[%d] = %p\n", cnt, tc+cnt);
      //(tc+cnt)->display();
      tc[cnt].index=cnt;
      if (cnt==MAX_PROBS-1) {
         printf("Computing %d testcases\n", cnt+1);
         fflush(0);
         ComputationTime.start();
         compute_full_prob_multiple(prob, tc, cnt+1, gmem);
         ComputationTime.acc();
         for (int q=0;q<cnt+1;q++) {
            //printf("%s vs %s\n", tc[q].rs, tc[q].hap);
		      printf("%E\n", q+basecnt, prob[q]);
         }
         cnt = -1;
         basecnt+=MAX_PROBS;
      }
      cnt++;

   }
   
   printf("Computing %d testcases\n", cnt);
   ComputationTime.start();
   if (cnt>0) compute_full_prob_multiple(prob, tc, cnt, gmem);
   ComputationTime.acc();

   //This call frees memory in gmem
   compute_full_prob_multiple(prob, tc, 0, gmem);

   for (int q=0;q<cnt;q++) {
            //printf("%s vs %s\n", tc[q].rs, tc[q].hap);
     printf("%E\n", q+basecnt, prob[q]);
   }

   TotalTime.acc();

   delete []tc;
   free(prob);
	return 0;
}


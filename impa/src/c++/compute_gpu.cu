#include "hip/hip_runtime.h"
//#include "input.h"
#include "compute_gpu.h"
#include "stdio.h"

__device__ double __shfl_up(double d,unsigned int i){
   double ret;
   ((float*)&ret)[0] = __shfl_up(((float*)&d)[0], i);
   ((float*)&ret)[1] = __shfl_up(((float*)&d)[1], i);
   return ret;
}

__device__ double __shfl_down(double d,unsigned int i){
   double ret;
   ((float*)&ret)[0] = __shfl_down(((float*)&d)[0], i);
   ((float*)&ret)[1] = __shfl_down(((float*)&d)[1], i);
   return ret;
}

template<class NUMBER>
__global__ void 
__launch_bounds__(128,8)
pairhmm_kernel( NUMBER Yr0, NUMBER* M, NUMBER *X, NUMBER *Y, 
                               NUMBER* p, char* rs, char* hap, 
                               NUMBER* q, int* offset, int n_mats,
                               NUMBER* output, NUMBER log10_init) {
   NUMBER M_p, M_pp, X_p, X_pp, Y_p, Y_pp, distm, pMM, pGapM, pXX, 
          pMX, pYY, pMY, M_loc, X_loc, Y_loc;
   char _rs;
   NUMBER _q;
   int tid = threadIdx.x;
   int wid = blockIdx.x;
   //Fake wid,tid to allow for larger blocks
   tid = (threadIdx.x + blockDim.x * blockIdx.x)%WARP;
   wid = (threadIdx.x + blockDim.x * blockIdx.x)/WARP;
   if (wid > n_mats) return;
   int ROWS = offset[3*wid+4]-offset[3*wid+1];
   int COLS = offset[3*wid+5]-offset[3*wid+2];
   NUMBER M_top, X_top, Y_top;
   NUMBER result=0.0;
   M+=offset[3*wid];
   X+=offset[3*wid];
   Y+=offset[3*wid];
   p+=6*offset[3*wid+1];
   rs+=offset[3*wid+1];
   hap+=offset[3*wid+2];
   q+=offset[3*wid+1];
   for (int stripe = 0; stripe < ROWS; stripe+=WARP-1) {
      if ( stripe==0 && tid < 2) {
         M_pp=0.0;
         X_pp = 0.0;//X_pp=Xc0[0];
         Y_pp=Yr0;
         M_p=0.0;
         if (tid==1) X_p = 0.0;//X_p=Xc0[1];
         else X_p=0.0;
         if (tid==0) Y_p=Yr0;
         else Y_p=0.0;
      } else if (tid<2) {
         M_pp = 0.0;
         X_pp = 0.0; //X_pp = Xc0[stripe]; //X[stripe][0];
         Y_pp = 0.0;
         M_p = 0.0;
         if (tid==1) X_p=0.0; //X_p=Xc0[1];
         else X_p = X[stripe/(WARP-1)*COLS+1]; //M[stripe+tid][1-tid];
         if (tid==0) Y_p = Y[stripe/(WARP-1)*COLS+1]; //M[stripe+tid][1-tid];
         else Y_p = 0.0;
      }
      //TODO pad instead
      if (tid>0) {
      _rs = rs[tid-1+stripe];
	   _q = q[tid-1+stripe];
      } else {
      _rs = rs[tid+stripe];
	   _q = q[tid+stripe];
      }
      //TODO transpose p for coalesced reads
      pMM = p[6*(tid+stripe)+MM];
      pGapM = p[6*(tid+stripe)+GapM];
      pXX = p[6*(tid+stripe)+XX];
      pMX = p[6*(tid+stripe)+MX];
      pYY = p[6*(tid+stripe)+YY];
      pMY = p[6*(tid+stripe)+MY];
      for (int z = 1; z < COLS+WARP+1;z++) 
      {
         int r = tid+stripe;
         int c = z-tid+1;
         //TODO align at word boundaries
         if (1==z%WARP) {
            M_top = M[stripe/(WARP-1)*COLS+z+tid];
            X_top = X[stripe/(WARP-1)*COLS+z+tid];
            Y_top = Y[stripe/(WARP-1)*COLS+z+tid];
         } else {
            M_top = __shfl_down(M_top,1);
            X_top = __shfl_down(X_top,1);
            Y_top = __shfl_down(Y_top,1);
         }

         if (tid<= z+1 && tid+stripe < ROWS && z-tid < COLS)
         {
            //TODO pad instead
            if (c>0) {
            char _hap = hap[c-1];
			   if (_rs == _hap || _rs == 'N' || _hap == 'N')
			   	distm = double(1.0) - _q;
            else distm = _q;
            }
            else distm = _q;
            if (tid == 0 && stripe == 0) {
               X_p = 0.0; 
               Y_p = Yr0;
               M_p = 0.0;
            } else if (tid == 0 && z > 1) {
               M_p = M_top;
               X_p = X_top;
               Y_p = Y_top;
            } 
            M_loc = distm * (M_pp * pMM + X_pp * pGapM + Y_pp * pGapM);
			   Y_loc = M_p * pMY + Y_p * pYY;
            M_p = __shfl_up(M_p,1);
            Y_p = __shfl_up(Y_p,1);
            X_p = __shfl_up(X_p,1);
			   X_loc = M_p * pMX + X_p * pXX;
            M_pp = M_p;
            X_pp = X_p;
            Y_pp = Y_p;
            if (tid == z+1 && stripe==0) {
               M_p = 0.0;
               Y_p = 0.0;
               X_p = 0.0; //X_p = Xc0[tid];
            } else if (tid == z+1) {
               M_p = 0.0;
               Y_p = 0.0;
               X_p = 0.0; //X_p = Xc0[tid+stripe]; //X[tid+stripe][0]
            } else {
               M_p = M_loc;
               X_p = X_loc;
               Y_p = Y_loc;
            }
            r = tid+stripe;
            c =  z-tid+1;
            //TODO shuffle M_out to write one time
            if (tid>0 && c < COLS && (r==stripe+WARP-1 || r==ROWS-1)) {
               M[((r+WARP-2)/(WARP-1))*COLS+c] = M_loc;
               X[((r+WARP-2)/(WARP-1))*COLS+c] = X_loc;
               Y[((r+WARP-2)/(WARP-1))*COLS+c] = Y_loc;
            }
            if (r==ROWS-1) { 
               result += M_loc + X_loc;
            }
         }
#if 0
         NUMBER M_bottom, X_bottom, Y_bottom;
         int write_tid=min(WARP-1, ROWS-stripe-1);
         //shuffle from write_tid to the last thread in the warp
         M_bottom = __shfl_down(M_bottom, 1);
         X_bottom = __shfl_down(X_bottom, 1);
         Y_bottom = __shfl_down(Y_bottom, 1);
         M_loc = __shfl_up(M_loc, WARP-1-write_tid); 
         X_loc = __shfl_up(X_loc, WARP-1-write_tid); 
         Y_loc = __shfl_up(Y_loc, WARP-1-write_tid); 
         if (tid == write_tid) {
            M_bottom = M_loc;
            X_bottom = X_loc;
            Y_bottom = Y_loc;
         }
         if (WARP-1 == (z-write_tid)%WARP || COLS-1 == z-write_tid) {
            r = stripe + write_tid;
            c = z - write_tid + 1;
            M[((r+WARP-2)/(WARP-1))*COLS+c-write_tid+tid] = M_bottom;
            X[((r+WARP-2)/(WARP-1))*COLS+c-write_tid+tid] = X_bottom;
            Y[((r+WARP-2)/(WARP-1))*COLS+c-write_tid+tid] = Y_bottom;
         }
#endif
         
      }
   }
   if (tid == (ROWS-1)%(WARP-1)) {
      output[wid] = log10(result) - log10_init; 
   }
   //if (tid == (ROWS-1)%(WARP-1)) output[wid] = result;
}
template<class NUMBER>
int GPUmemAlloc(GPUmem<NUMBER>& gmem) 
{
   hipDeviceProp_t deviceProp;
   hipError_t err = hipGetDeviceProperties(&deviceProp, 0);
   unsigned long long totalMem = deviceProp.totalGlobalMem/5;
   //TODO 
   //allocations based (very loosely) on ROWS=COLS=10
   //TODO readjust sizes
   hipMalloc(&gmem.d_M, totalMem/10);
   gmem.M = (NUMBER*)malloc(totalMem/10);
   hipMalloc(&gmem.d_X, totalMem/10);
   gmem.X = (NUMBER*)malloc(totalMem/10);
   hipMalloc(&gmem.d_Y, totalMem/10);
   gmem.Y = (NUMBER*)malloc(totalMem/10);
   hipMalloc(&gmem.d_p, totalMem/11);
   gmem.p = (NUMBER*)malloc(totalMem/11);
   hipMalloc(&gmem.d_Yr0, totalMem/66);
   gmem.Yr0 = (NUMBER*)malloc(totalMem/66);
   hipMalloc(&gmem.d_Xc0, totalMem/66);
   gmem.Xc0 = (NUMBER*)malloc(totalMem/66);
   hipMalloc(&gmem.d_q, totalMem/66);
   gmem.q = (NUMBER*)malloc(totalMem/66);
   hipMalloc(&gmem.d_rs, 700000);
   gmem.rs = (char*)malloc(700000);
   hipMalloc(&gmem.d_hap, 700000);
   gmem.hap = (char*)malloc(700000);
   err = hipGetLastError();
   if (err) printf("hipMalloc error %d: %s\n", err, hipGetErrorString(err));
   if (err) return 9000+err;
   if (!gmem.M ||
       !gmem.X ||
       !gmem.Y ||
       !gmem.p ||
       !gmem.q ||
       !gmem.rs ||
       !gmem.Yr0 ||
       !gmem.Xc0 ||
       !gmem.hap) {
      printf("CPU mem allocation fail\n");
      return 1;
   }
   return 0;
}
template<class NUMBER>
int GPUmemFree(GPUmem<NUMBER>& gmem) 
{
   if (0==gmem.M) {
      return 0;
   }
   gmem.index=0;
   hipFree(gmem.d_M);
   free(gmem.M);
   gmem.M=0;
   hipFree(gmem.d_X);
   free(gmem.X);
   gmem.X=0;
   hipFree(gmem.d_Y);
   free(gmem.Y);
   gmem.Y=0;
   hipFree(gmem.d_p);
   free(gmem.p);
   gmem.p=0;
   hipFree(gmem.d_Yr0);
   free(gmem.Yr0);
   gmem.Yr0=0;
   hipFree(gmem.d_Xc0);
   free(gmem.Xc0);
   gmem.Xc0=0;
   hipFree(gmem.d_q);
   free(gmem.q);
   gmem.q=0;
   hipFree(gmem.d_rs);
   free(gmem.rs);
   gmem.rs=0;
   hipFree(gmem.d_hap);
   free(gmem.hap);
   gmem.hap=0;
   return 0;
}
template int GPUmemAlloc<double>(GPUmem<double>&);
template int GPUmemAlloc<float>(GPUmem<float>&);
template int GPUmemFree<double>(GPUmem<double>&);
template int GPUmemFree<float>(GPUmem<float>&);

template <class PRECISION>
void compute_gpu(int offset[][3], PRECISION* p, char* rs, char* hap, PRECISION* q, 
                           PRECISION Yr0, int n_tc, PRECISION* h_out, GPUmem<PRECISION>& gmem) 
{
   //GPUmem<PRECISION> gmem;
   PRECISION *d_out;
   hipError_t cuerr;

   if (0==gmem.M) {
      GPUmemAlloc<PRECISION>(gmem);
   }
   hipMalloc(&d_out,  sizeof(PRECISION)*n_tc);
   hipMalloc(&gmem.d_offset, sizeof(int)*3*(n_tc+1));
   hipMemcpy(gmem.d_offset, &offset[0][0], sizeof(int)*3*(n_tc+1), hipMemcpyHostToDevice);
   hipMemcpy(gmem.d_p, p, sizeof(PRECISION)*offset[n_tc][1]*6, hipMemcpyHostToDevice);
   hipMemcpy(gmem.d_rs, rs, sizeof(char)*offset[n_tc][1], hipMemcpyHostToDevice);
   hipMemcpy(gmem.d_hap, hap, sizeof(char)*offset[n_tc][2], hipMemcpyHostToDevice);
   hipMemcpy(gmem.d_q, q, sizeof(PRECISION)*offset[n_tc][1], hipMemcpyHostToDevice);
   fflush(0);
   cuerr= hipGetLastError();
   if (cuerr) printf("Error in memcpy. %d : %s\n", cuerr, hipGetErrorString(cuerr));
   //One warp handles one matrix
	PRECISION INITIAL_CONSTANT = ldexp(1.0, 1020.0);
	PRECISION LOG10_INITIAL_CONSTANT = log10(INITIAL_CONSTANT);
   
   pairhmm_kernel<<<(n_tc+3)/4,WARP*4>>>( Yr0, gmem.d_M, gmem.d_X, 
                                  gmem.d_Y, gmem.d_p, 
                                  gmem.d_rs, gmem.d_hap, gmem.d_q,
                                  gmem.d_offset, n_tc-1, d_out, LOG10_INITIAL_CONSTANT); 
   cuerr = hipGetLastError();
   if (cuerr) {
      printf ("Cuda error %d : %s\n", cuerr, hipGetErrorString(cuerr));
   }
#if 0
   PRECISION *M2;
   PRECISION *X2;
   //TODO Fix this!
   hipMemcpy(gmem.M, gmem.d_M,
                  sizeof(PRECISION)*gmem.offset[n_tc][0]+1338,
                  hipMemcpyDeviceToHost);
   hipMemcpy(gmem.X, gmem.d_X,
                  sizeof(PRECISION)*gmem.offset[n_tc][0]+1338,
                  hipMemcpyDeviceToHost);
   cuerr = hipGetLastError();
   if (cuerr) printf ("Memcpy(D2H) error %d : %s\n", cuerr, hipGetErrorString(cuerr));
   for (int z=0;z<n_tc;z++)
   {
      int ROWS = gmem.offset[z+1][1]-gmem.offset[z][1];
      int COLS = gmem.offset[z+1][2]-gmem.offset[z][2];
      M2 = gmem.M+gmem.offset[z][0]+COLS*(((ROWS-1)+WARP-2)/(WARP-1));
      X2 = gmem.X+gmem.offset[z][0]+COLS*(((ROWS-1)+WARP-2)/(WARP-1));
	   PRECISION result = 0.0;
	   for (int c = 0; c < COLS; c++)
	   	result += M2[c] + X2[c];

   	if (before_last_log != NULL)
   		*before_last_log = result;	

      probs[z] = log10(out[z]) - LOG10_INITIAL_CONSTANT;
   }
#else
   hipMemcpy(h_out, d_out, sizeof(PRECISION)*n_tc, hipMemcpyDeviceToHost);
#endif
   hipFree(d_out);
   //GPUmemFree(gmem);
}
template void compute_gpu<double>(int [][3], double*, char*, char*, double*, double, int, double*, GPUmem<double>&);
template void compute_gpu<float>(int [][3], float*, char*, char*, float*, float, int, float*, GPUmem<float>&);
